#include "cudadevice.h"
#include <cassert>
#include <hip/hip_runtime.h>
#include <sstream>

int CudaDevice::deviceCount()
{
    int n;
    hipError_t res;
    
    res = hipGetDeviceCount(&n);
    if (res != hipSuccess) { n = 0; }
    return n;
}

CudaDevice::CudaDevice(int dev)
{
    hipError_t res;
    m_prop = new hipDeviceProp_t;
    
    res = hipGetDeviceProperties(m_prop, dev);
    if (res != hipSuccess) {
        std::stringstream sstr;
        
        delete m_prop;
        sstr << "CudaDevice: invalid device " << dev << '.';
        throw Exception(sstr.str());
    }
}

CudaDevice::~CudaDevice()
{
    delete m_prop;
}

// hide default copy constructor
CudaDevice::CudaDevice(const CudaDevice&)
{
    assert(false);
}

// hide default assigment
CudaDevice& CudaDevice::operator = (const CudaDevice&)
{
    assert(false);
    return *this;
}

int CudaDevice::majorVersion() const
{
    return m_prop -> major;
}

int CudaDevice::minorVersion() const
{
    return m_prop -> minor;
}

const char* CudaDevice::name() const
{
    return m_prop -> name;
}

unsigned CudaDevice::totalGlobalMem() const
{
    return m_prop -> totalGlobalMem;
}

unsigned CudaDevice::totalConstMem() const
{
    return m_prop -> totalConstMem;
}

unsigned CudaDevice::sharedMemPerBlock() const
{
    return m_prop -> sharedMemPerBlock;
}

int CudaDevice::maxThreadsPerBlock() const
{
    return m_prop -> maxThreadsPerBlock;
}

int CudaDevice::maxThreadsPerMultiProcessor() const
{
    return m_prop -> maxThreadsPerMultiProcessor;
}

int CudaDevice::maxGridSizeX() const
{
    return m_prop -> maxGridSize[0];
}

int CudaDevice::maxGridSizeY() const
{
    return m_prop -> maxGridSize[1];
}

int CudaDevice::maxGridSizeZ() const
{
    return m_prop -> maxGridSize[2];
}

int CudaDevice::maxThreadsDimX() const
{
    return m_prop -> maxThreadsDim[0];
}

int CudaDevice::maxThreadsDimY() const
{
    return m_prop -> maxThreadsDim[1];
}

int CudaDevice::maxThreadsDimZ() const
{
    return m_prop -> maxThreadsDim[2];
}

int CudaDevice::warpSize() const
{
    return m_prop -> warpSize;
}

bool CudaDevice::deviceOverlap() const
{
    return m_prop -> deviceOverlap > 0;
}

int CudaDevice::asyncEngineCount() const
{
    return m_prop -> asyncEngineCount;
}

bool CudaDevice::deviceMapHost() const
{
    return m_prop -> canMapHostMemory > 0;
}
