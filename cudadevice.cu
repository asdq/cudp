#include <hip/hip_runtime.h>
#include <cstdio>
#include "cudadevice.h"

#define cudaCheck(stmt)                                     \
    do {                                                    \
        hipError_t err = stmt;                             \
        if (err != hipSuccess) {                           \
            fprintf(stderr, "%s in %s at line %d\n",        \
            hipGetErrorString(err), __FILE__, __LINE__);   \
            hipDeviceReset();                              \
            exit(EXIT_FAILURE);                             \
        }                                                   \
    } while(0)

int CudaDevice::deviceCount()
{
    int n;
    
    cudaCheck(hipGetDeviceCount(&n));
    return n;
}

CudaDevice::CudaDevice(int dev)
{
    auto p = new hipDeviceProp_t;
    cudaCheck(hipGetDeviceProperties(p, dev));
    m_prop = std::unique_ptr<hipDeviceProp_t>(p);
}

CudaDevice::~CudaDevice() {}

int CudaDevice::majorVersion() const
{
    return m_prop -> major;
}

int CudaDevice::minorVersion() const
{
    return m_prop -> minor;
}

char* CudaDevice::name() const
{
    return m_prop -> name;
}

unsigned CudaDevice::totalGlobalMem() const
{
    return m_prop -> totalGlobalMem;
}

unsigned CudaDevice::totalConstMem() const
{
    return m_prop -> totalConstMem;
}

unsigned CudaDevice::sharedMemPerBlock() const
{
    return m_prop -> sharedMemPerBlock;
}

int CudaDevice::maxThreadsPerBlock() const
{
    return m_prop -> maxThreadsPerBlock;
}

int CudaDevice::maxThreadsPerMultiProcessor() const
{
    return m_prop -> maxThreadsPerMultiProcessor;
}

int CudaDevice::maxGridSizeX() const
{
    return m_prop -> maxGridSize[0];
}

int CudaDevice::maxGridSizeY() const
{
    return m_prop -> maxGridSize[1];
}

int CudaDevice::maxGridSizeZ() const
{
    return m_prop -> maxGridSize[2];
}

int CudaDevice::maxThreadsDimX() const
{
    return m_prop -> maxThreadsDim[0];
}

int CudaDevice::maxThreadsDimY() const
{
    return m_prop -> maxThreadsDim[1];
}

int CudaDevice::maxThreadsDimZ() const
{
    return m_prop -> maxThreadsDim[2];
}

int CudaDevice::warpSize() const
{
    return m_prop -> warpSize;
}

bool CudaDevice::deviceOverlap() const
{
    return m_prop -> deviceOverlap > 0;
}

int CudaDevice::asyncEngineCount() const
{
    return m_prop -> asyncEngineCount;
}

bool CudaDevice::deviceMapHost() const
{
    return m_prop -> canMapHostMemory;
}
